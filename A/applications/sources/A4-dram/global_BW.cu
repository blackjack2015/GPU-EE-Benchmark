#include "hip/hip_runtime.h"
/*
 *
 * globalCopy.cu
 *
 * Microbenchmark for copy bandwidth of global memory.
 *
 * Build with: nvcc -I ../chLib <options> globalCopy.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "sys/time.h"

//SM number: 8(Fermi 560 Ti); 12(Kepler 780 ); 16 (Maxwell 980); 28 (Pascal Titan X); 80 (Tesla V100)
#define BLOCK_BASE (80)
#define MULTIPLIER (10)

template<const int n> 
__global__ void GlobalCopy(int *out, const int *in, size_t N )
{
    int temp[n];

	//avoid accessing cache, assure cold-cache access
	int start = n * blockIdx.x * blockDim.x + threadIdx.x;
	int step = n * blockDim.x * gridDim.x;
    
	int i;

    for ( i = start; i < N - step; i += step ) {
        for ( int j = 0; j < n; j++ ) {
            int index = i+j*blockDim.x;
            temp[j] = in[index];
        }
        for ( int j = 0; j < n; j++ ) {
            int index = i+j*blockDim.x;
            out[index] = temp[j];
        }
    }
    //there may be some elements left due to misaligning.
    for ( int j = 0; j < n; j++ ) {
        for ( int j = 0; j < n; j++ ) {
			int index = i + j*blockDim.x;
            if ( index<N ) temp[j] = in[index];
        }
        for ( int j = 0; j < n; j++ ) {
			int index = i + j*blockDim.x;
            if ( index<N ) out[index] = temp[j];
        }
    }
}

template<const int n>
double BandwidthCopy( int *deviceOut, int *deviceIn,
               int *hostOut, int *hostIn,
               size_t N,
               int cBlocks, int cThreads )
{
    double ret = 0.0;
    double elapsedTime;
    int cIterations;
    hipError_t status;

    for ( int i = 0; i < N; i++ ) {
        int r = rand();
		hostIn[i] = *(int *)(&r); // for small ints, LSBs; for int2 and int4, some stack cruft
    }

	memset(hostOut, 0, N*sizeof(int));
	hipMemcpy(deviceIn, hostIn, N*sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
    {
        // confirm that kernel launch with this configuration writes correct result
        GlobalCopy<n><<<cBlocks,cThreads>>>( 
            deviceOut,
            deviceIn,
            N );
		hipDeviceSynchronize();
		hipMemcpy(hostOut, deviceOut, N*sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        status = hipGetLastError() ; 
		if (memcmp(hostOut, hostIn, (N)*sizeof(int))) {
            printf( "Incorrect copy performed!\n" );
            return -1.0;
        }
    }

    int secs = -1;
    cIterations = 10;
    struct timeval start, end;

    // Get environment variables
    if (getenv("secs") != NULL)
        secs = atoi(getenv("secs"));

	//time_t start, end;
	//start = clock();
    double total_time = 0;
    for ( int i = -10; i < cIterations; i++ ) {
        gettimeofday(&start,NULL);
	GlobalCopy<n> << <cBlocks, cThreads >> >(deviceOut, deviceIn, N);
        hipDeviceSynchronize();
        gettimeofday(&end,NULL);
	status = hipGetLastError();

        total_time += end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec)/1000000.0;

        if (i == -1){
            if (secs > 0){
                double estimated_time = total_time / 10.0;
                cIterations = int((double)secs / estimated_time) + 1;
            }
            total_time = 0;
        }
    }

    // make configurations that cannot launch error-out with 0 bandwidth
    //end = clock();

    //elapsedTime =  (end - start)/1000.0;
    elapsedTime =  total_time;

    // bytes per second
	ret = ((double)2 * N*cIterations*sizeof(int)) / elapsedTime;
    // gigabytes per second
    ret /= 1024.0*1048576.0;
    return ret;
}

template<const int n>
double ReportRow( size_t N, size_t threadStart, size_t threadStop, size_t cBlocks)
{
    int *deviceIn = 0;
    int *deviceOut = 0;
    int *hostIn = 0;
    int *hostOut = 0;

    hipError_t status;

    int maxThreads = 0;
    double maxBW = 0.0;

	hipMalloc(&deviceIn, N*sizeof(int));
	hipMalloc(&deviceOut, N*sizeof(int));
	hipMemset(deviceOut, 0, N*sizeof(int));

	hostIn = new int[N];
	hostOut = new int[N];
	if (!hostIn || !hostOut){
		if (hostIn) delete[] hostIn;
		if (hostOut) delete[] hostOut;

		hipFree(deviceIn);
		hipFree(deviceOut);
		return maxBW;
	}

    printf( "%d\t", n );

    for ( int cThreads = threadStart; cThreads <= threadStop; cThreads *= 2 ) {
        double bw = BandwidthCopy<n>(
            deviceOut, deviceIn, hostOut, hostIn, N,
             cBlocks, cThreads );
        if ( bw > maxBW ) {
            maxBW = bw;
            maxThreads = cThreads;
        }
        printf( "%.2f\t", bw );
    }
    printf( "%.2f\t%d\n", maxBW, maxThreads );

	delete[] hostIn;
	delete[] hostOut;

	hipFree(deviceIn);
	hipFree(deviceOut);
	return maxBW;
}

int main()
{

    // Get environment variables
    int device = 0;
    if (getenv("device") != NULL)
        device = atoi(getenv("device"));

    int size = 64;
	int N = size * 1048576;
	int block_num;
	int threadStart = 512;
	int threadStop = 1024;

    printf( "Using coalesced reads and writes\n" );
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);

        double maxBW = 0.0;
        double retBW = 0.0;
	// for ( block_num =int(BLOCK_BASE); block_num <= int(BLOCK_BASE * MULTIPLIER); block_num += int(BLOCK_BASE)){
	for ( block_num =int(BLOCK_BASE); block_num <= int(BLOCK_BASE); block_num += int(BLOCK_BASE)){
		printf("\n=================Block number: %d=================\n", block_num);
		printf("Operand size: %d byte%c\n", sizeof(int), sizeof(int) == 1 ? '\0' : 's');
		printf("Input size: %dM operands\n", (int)(N >> 20));
		printf("                      Block Size\n");
		printf("Unroll\t");

		for (int cThreads = threadStart; cThreads <= threadStop; cThreads *= 2) {
			printf("%d\t", cThreads);
		}

		printf("maxBW\tmaxThreads\n");
		retBW = ReportRow<4>(N, threadStart, threadStop, block_num);
                if (retBW > maxBW)
                    maxBW = retBW;

		retBW = ReportRow<8>(N, threadStart, threadStop, block_num);
                if (retBW > maxBW)
                    maxBW = retBW;

	}

    double maxTHR = (N * 2 * 1.0) / (2 * N * sizeof(int) * 1.0 / maxBW);
    printf("Maximum bandwidth is %.3f GB/s.\n", maxBW);
    printf("Maximum throughput is %.3f GOP/s.\n", maxTHR);

    return 0;
}
