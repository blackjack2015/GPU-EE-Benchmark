#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define VECTOR_SIZE 655360
#define TILE_DIM 1024
#define COMP_ITERATIONS 1024000

__global__ void simpleKernel(double *A, double *C1, int size, int compute_iters, int tile_dim)
{
    int xIndex = blockIdx.x * tile_dim + threadIdx.x;
    double ra, rc;

    if (xIndex < size) {
        ra=A[xIndex];
        rc=A[size - xIndex - 1];
        // rb=A[xIndex];
        for (int i=0;i<compute_iters;i++) {
            //add_2regs
            rc += ra * ra;
        }
        C1[xIndex]=rc;
    }
}


int main(int argc, char **argv) {
    int compute_iters=COMP_ITERATIONS,
        vector_size=VECTOR_SIZE,
        tile_dim=TILE_DIM;

    // execution configuration parameters
    dim3 grid(vector_size/tile_dim, 1), threads(tile_dim, 1);

    // CUDA events
    hipEvent_t start, stop;

    // allocate host memory
    size_t item_size = sizeof(double);
    size_t mem_size = item_size * vector_size;
    double *h_iA = (double *) malloc(mem_size);
    double *h_oC1 = (double *) malloc(mem_size);
    double elem = 2.7;
    // initalize host data
    for (int i = 0; i < vector_size; ++i)
    {
        // h_iA[i] = (float) i+3;
        h_iA[i] = elem;
    }
    // allocate device memory
    double *d_iA, *d_oC1;
    hipMalloc((void **) &d_iA, mem_size);
    hipMalloc((void **) &d_oC1, mem_size);

    // copy host data to device
    hipMemcpy(d_iA, h_iA, mem_size, hipMemcpyHostToDevice);

    // print out common data for all kernels
    printf("\nVector size: %d  TotalBlocks: %d blockSize: %d\n\n", vector_size, grid.x, threads.x);

    // initialize events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int secs = -1;
    int cIterations = 10;

    // Get environment variables
    if (getenv("secs") != NULL)
        secs = atoi(getenv("secs"));

    double total_time = 0;
    float kernelTime;
    for(int i = -10; i < cIterations; i++){
        hipEventRecord(start, 0);
        simpleKernel<<<grid, threads>>>(d_iA, d_oC1, vector_size, compute_iters, tile_dim);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&kernelTime, start, stop);

        total_time += kernelTime / 1000.0;
        if (i == -1){
            if (secs > 0){
                double estimated_time = total_time / 10.0;
                cIterations = int((double)secs / estimated_time) + 1;
                printf("Estimated second is %f, adjust iteration to %d.\n", estimated_time, cIterations);
            }
            total_time = 0;
        }
    }

    kernelTime = total_time / cIterations;

    // take measurements for loop inside kernel
    hipMemcpy(h_oC1, d_oC1, mem_size, hipMemcpyDeviceToHost);

    printf("teste: %f\n", h_oC1[0]);

    //float peak_bw = 2 * compute_iters * mem_size * 1.0 / kernelTime / (1024.*1024.*1024.); 
    float peak_bw = 2 * compute_iters * mem_size * 1.0 / kernelTime / (1024.*1024.*1024.); 
    printf("Maximum bandwidth is %.3f GB/s.\n", peak_bw);
    printf("Maximum throughput is %.3f GOP/s.\n", peak_bw / item_size);

    free(h_iA);
    free(h_oC1);

    hipFree(d_iA);
    hipFree(d_oC1);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();

    printf("Test passed\n");

    exit(EXIT_SUCCESS);
}
